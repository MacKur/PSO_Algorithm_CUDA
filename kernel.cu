#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernel.h"


__device__ float particle_1[DIMENSIONS];
__device__ float particle_2[DIMENSIONS];


__device__ float dev_rosenbrock_function(float x[])
{
   float fitness = 0;
   for (int c = 0; c < DIMENSIONS - 1; c++)
   {	
	fitness += 100 * pow(x[c + 1] - (x[c] * x[c]), 2) + pow(1 - x[c], 2);
   }
   return fitness;
}


__device__ float dev_ackleys_function(float x[])
{
   float first_sum = 0;
   float second_sum = 0;
   for (int c = 0; c < DIMENSIONS; c++)
   {
	first_sum += pow(x[c], 2);
	second_sum += cos(2.0 * phi * x[c]);
   }   
   float fitness = -20 * exp(-0.2 * sqrt(first_sum / 2)) - exp(second_sum / 2) + 20 + 2.7183;
   return fitness;
}


__device__ float dev_rastrigin_function(float x[])
{
   float fitness = 20;
   for (int c = 0; c < DIMENSIONS; c++)
   {
	fitness += pow(x[c], 2) - (10 * cos(2.0 * phi * x[c]));
   }
   return fitness;
}


__device__ float dev_sphere_function(float x[])
{
   float fitness = 0;
   for (int c = 0; c < DIMENSIONS; c++)
   {
	fitness += pow(x[c], 2);
   }
   return fitness;
}


__global__ void kernelUpdateParticle(float *particle_position, float *particle_velocity, 
                                     float *personal_best_position, float *global_best_position, float r1, 
                                     float r2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= PARTICLES * DIMENSIONS)
        return;
    
    float rp = r1;
    float rg = r2;

    particle_velocity[i] = w * particle_velocity[i] + c1 * rp * (personal_best_position[i] - particle_position[i])
            + c2 * rg * (global_best_position[i % DIMENSIONS] - particle_position[i]);

    particle_position[i] = particle_position[i] +  particle_velocity[i];
}

__global__ void kernelUpdatePersonalBest(float *particle_position, float *personal_best_position, float *global_best_position)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= PARTICLES * DIMENSIONS || i % DIMENSIONS != 0)
        return;

    for (int j = 0; j < DIMENSIONS; j++)
    {
        particle_1[j] = particle_position[i + j];
        particle_2[j] = personal_best_position[i + j];
    }

    if (dev_rosenbrock_function(particle_1) < dev_rosenbrock_function(particle_2))
    {
        for (int k = 0; k < DIMENSIONS; k++)
            personal_best_position[i + k] = particle_position[i + k];
    }
}


extern "C" void pso_gpu(float *particle_position, float *particle_velocity, float *personal_best_position, 
                         float *global_best_position)
{
    int size = PARTICLES * DIMENSIONS;
    
    // Wskaźniki
    float *devPosition;
    float *devVelocity;
    float *devPersonalBest;
    float *devGlobalBest;
    
    float temp[DIMENSIONS];
        
    // Alokacja pamięci
    hipMalloc((void**)&devPosition, sizeof(float) * size);
    hipMalloc((void**)&devVelocity, sizeof(float) * size);
    hipMalloc((void**)&devPersonalBest, sizeof(float) * size);
    hipMalloc((void**)&devGlobalBest, sizeof(float) * DIMENSIONS);
    
    // Liczba wątków i bloków
    int threadsNum = 32;
    int blocksNum = PARTICLES / threadsNum; // (512 / 32) = 16
    
    // Pobranie cząsteczek z hosta na dev
    hipMemcpy(devPosition, particle_position, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devVelocity, particle_velocity, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devPersonalBest, personal_best_position, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devGlobalBest, global_best_position, sizeof(float) * DIMENSIONS, hipMemcpyHostToDevice);
    
    // PSO
    for (int iter = 0; iter < ITERATIONS; iter++)
    {     
        // Aktualizacja pozycji i prędkości cząsteczek
        kernelUpdateParticle<<<blocksNum, threadsNum>>>(devPosition, devVelocity, 
                                                        devPersonalBest, devGlobalBest, 
                                                        getRandomLimited(), 
                                                        getRandomLimited());  
        // Aktualizacja najlepszej cząsteczki (lokalnej)
        kernelUpdatePersonalBest<<<blocksNum, threadsNum>>>(devPosition, devPersonalBest, devGlobalBest);
        
        // Aktualizacja najlepszej cząsteczki globalnie dla całej populacji
        hipMemcpy(personal_best_position, devPersonalBest, 
                   sizeof(float) * PARTICLES * DIMENSIONS, 
                   hipMemcpyDeviceToHost);
        
        for(int i = 0; i < size; i += DIMENSIONS)
        {
            for(int k = 0; k < DIMENSIONS; k++)
                temp[k] = personal_best_position[i + k];
        
            if (rosenbrock_function(temp) < rosenbrock_function(global_best_position))
            {
                for (int k = 0; k < DIMENSIONS; k++)
                    global_best_position[k] = temp[k];
            }   
        }
        
        hipMemcpy(devGlobalBest, global_best_position, sizeof(float) * DIMENSIONS, 
                   hipMemcpyHostToDevice);
    }
    
    // Pobranie cząsteczek z dev na hosta
    hipMemcpy(particle_position, devPosition, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(particle_velocity, devVelocity, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(personal_best_position, devPersonalBest, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(global_best_position, devGlobalBest, sizeof(float) * DIMENSIONS, hipMemcpyDeviceToHost); 
    
    
    hipFree(devPosition);
    hipFree(devVelocity);
    hipFree(devPersonalBest);
    hipFree(devGlobalBest);
}
